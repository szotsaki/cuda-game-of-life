#include "hip/hip_runtime.h"
#include <cstdio>
#include <cstdlib>
#include <ctime>
#include <iostream>
#include <algorithm>

#include "GameOfLife.h"

/**
 * This macro checks return value of the CUDA runtime call and exits
 * the application if the call failed.
 */
#define CUDA_CHECK_RETURN(value) {											\
	hipError_t _m_cudaStat = value;										\
	if (_m_cudaStat != hipSuccess) {										\
		fprintf(stderr, "Error: %s at line %d in file %s\n",					\
				hipGetErrorString(_m_cudaStat), __LINE__, __FILE__);		\
		exit(1);															\
	} }

__device__ short getLivingNeighbouringCells(bool state[], const int phase, const int index, const int width, const int height) {
    short livingCells = 0;
    const int offset = phase * width * height;
    int icLeft = 0, icRight = 0, icBottom = 0, icTop = 0;

    // Creating an infinite field
    if ((index % width) == 0) { // On the left side
        icLeft = width;
    } else if ((index % (width - 1)) == 0) { // On the right side
        icRight = -(width - 1);
    }
    if (index >= 0 && index <= width) { // On the top
        icTop = (height - 1) * width;
    } else if (index >= (width * height) - width && index <= width * height) { // On the bottom
        icBottom = -((height - 1) * width);
    }

    livingCells += state[offset + index - 1 + icLeft] + state[offset + index + 1 + icRight];
    livingCells += state[offset + index - width + icTop] + state[offset + index - width - 1 + icLeft + icTop] + state[offset + index - width + 1 + icRight + icTop];
    livingCells += state[offset + index + width + icBottom] + state[offset + index + width - 1 + icLeft + icBottom] + state[offset + index + width + 1 + icRight + icBottom];

    return livingCells;
}

__global__ void simulateGameOfLife(bool states[], const int steps, const int start, const int width, const int height) {
    const int i = blockDim.x * blockIdx.x + threadIdx.x;
    const int N = width * height;

    if (i >= N) {
        return;
    }

    for (int j = start; j < steps - 1; ++j) { // less and not lte, because we compute j+1
        const short cells = getLivingNeighbouringCells(states, j, i, width, height);
        if (cells == 2) {
            states[N * (j + 1) + i] = states[(N * j) + i];
        } else if (cells < 2 || cells > 3) {
            states[N * (j + 1) + i] = false;
        } else {
            states[N * (j + 1) + i] = true;
        }
    }
}

/**
 * Host function that prepares data array and passes it to the CUDA kernel.
 */
int main(void) {
    const int width = 600;
    const int height = 600;
    const int steps = 1500;
    const int totalStreams = 5;

    // Computed values
    const int N = width * height;
    const int threadsPerBlock = 128;
    const int blocksPerGrid = (N + threadsPerBlock - 1) / threadsPerBlock;
    const int stepsForOneKernel = steps / totalStreams;
    const size_t copySizeForOneKernel = stepsForOneKernel * N * sizeof(bool);

    std::clog << "Threads per block: " << threadsPerBlock << std::endl;
    std::clog << "Blocks per grid: " << blocksPerGrid << std::endl;
    std::clog << "Steps for a kernel: " << stepsForOneKernel << std::endl;

    hipStream_t streams[totalStreams];
    hipEvent_t events[totalStreams];
    for (int i = 0; i < totalStreams; ++i) {
        CUDA_CHECK_RETURN(hipStreamCreate(&streams[i]));
        CUDA_CHECK_RETURN(hipEventCreateWithFlags(&events[i], hipEventDisableTiming));
    }

    // Device allocation
    bool *d_states;
    size_t totalSize = N * sizeof(bool) * steps;
    CUDA_CHECK_RETURN(hipMalloc(&d_states, totalSize));

    // Host allocation
    bool *h_states;
    CUDA_CHECK_RETURN(hipHostMalloc(&h_states, totalSize));
    createRandomCells(h_states, width, height);

    // Host to device copy the first random step
    size_t sizeFirstStep = width * height * sizeof(bool);
    CUDA_CHECK_RETURN(hipMemcpy(d_states, h_states, sizeFirstStep, hipMemcpyHostToDevice));

    // Measure timing
    hipEvent_t cu_start, cu_end;
    float elapsedTime;
    CUDA_CHECK_RETURN(hipEventCreateWithFlags(&cu_start, hipEventBlockingSync));
    CUDA_CHECK_RETURN(hipEventCreateWithFlags(&cu_end, hipEventBlockingSync));
    CUDA_CHECK_RETURN(hipEventRecord(cu_start, 0));

    for (int i = 0; i < totalStreams; ++i) {
        const int start = i * stepsForOneKernel;
        const size_t offset = start * sizeof(bool);

        if (i > 0) {
            CUDA_CHECK_RETURN(hipEventSynchronize(events[i - 1]));
        }

        simulateGameOfLife<<<blocksPerGrid, threadsPerBlock, 0, streams[i]>>>(d_states, (i + 1) * stepsForOneKernel, start , width, height);
        CUDA_CHECK_RETURN(hipEventRecord(events[i], streams[i]));
        CUDA_CHECK_RETURN(hipMemcpyAsync(h_states + offset, d_states + offset, copySizeForOneKernel, hipMemcpyDeviceToHost, streams[i]));
    }

    CUDA_CHECK_RETURN(hipEventRecord(cu_end, 0));
    CUDA_CHECK_RETURN(hipEventSynchronize(cu_end));
    CUDA_CHECK_RETURN(hipEventElapsedTime(&elapsedTime, cu_start, cu_end));

    std::cout << (elapsedTime / 1000) << " seconds were needed to compute this Game of Life table" << std::endl;

    writeResultToFile("/home/aki/temp/gol.lif", h_states, width, height, steps);

    // Freeing resources
    CUDA_CHECK_RETURN(hipFree(d_states));
    CUDA_CHECK_RETURN(hipHostFree(h_states));
    CUDA_CHECK_RETURN(hipEventDestroy(cu_start));
    CUDA_CHECK_RETURN(hipEventDestroy(cu_end));
    CUDA_CHECK_RETURN(hipDeviceReset());

    return 0;
}
